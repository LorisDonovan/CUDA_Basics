#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include "book.h"
#include "cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

// texture references must be global
texture<float> texConstSrc;
texture<float> texInSrc;
texture<float> texOutSrc;

struct DataBlock
{
	uint8_t* outBitmap;
	float* d_InSrc;
	float* d_OutSrc;
	float* d_ConstSrc;
	CPUAnimBitmap* bitmap;

	hipEvent_t start, stop;
	float totalTime;
	float frames;
};

__global__ void CopyConstKernel(float* dst)
{
	int32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	int32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	int32_t offset = x + y * blockDim.x * gridDim.x;

	float c = tex1Dfetch(texConstSrc, offset);
	if (c != 0) // to preserve previous values
		dst[offset] = c;
}

__global__ void BlendKernel(float* dst, bool dstOut)
{
	int32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	int32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	int32_t offset = x + y * blockDim.x * gridDim.x;

	int32_t left = offset - 1;
	int32_t right = offset + 1;
	if (x == 0)
		left++;
	if (x == DIM - 1)
		right--;

	int32_t top = offset - DIM;
	int32_t bottom = offset + DIM;
	if (y == 0)
		top += DIM;
	if (y == DIM - 1)
		bottom -= DIM;

	float t, l, c, r, b;
	if (dstOut) // dstOut indicates whether to use buffer as input or output
	{
		t = tex1Dfetch(texInSrc, top);
		l = tex1Dfetch(texInSrc, left);
		c = tex1Dfetch(texInSrc, offset);
		r = tex1Dfetch(texInSrc, right);
		b = tex1Dfetch(texInSrc, bottom);
	}
	else
	{
		t = tex1Dfetch(texOutSrc, top);
		l = tex1Dfetch(texOutSrc, left);
		c = tex1Dfetch(texOutSrc, offset);
		r = tex1Dfetch(texOutSrc, right);
		b = tex1Dfetch(texOutSrc, bottom);
	}

	dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

void AnimGPU(DataBlock* d, int ticks)
{
	hipEventRecord(d->start, nullptr);
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	CPUAnimBitmap* bitmap = d->bitmap;

	volatile bool dstOut = true;
	for (int i = 0; i < 90; i++)
	{
		float* inSrc, * outSrc;
		if (dstOut)
		{
			inSrc = d->d_InSrc;
			outSrc = d->d_OutSrc;
		}
		else
		{
			outSrc = d->d_InSrc;
			inSrc = d->d_OutSrc;
		}

		CopyConstKernel << <blocks, threads >> > (inSrc);
		BlendKernel << <blocks, threads >> > (outSrc, dstOut);
		dstOut = !dstOut;
	}

	float_to_color << <blocks, threads >> > (d->outBitmap, d->d_InSrc);
	hipMemcpy(bitmap->get_ptr(), d->outBitmap, bitmap->image_size(), hipMemcpyDeviceToHost);

	hipEventRecord(d->stop, nullptr);
	hipEventSynchronize(d->stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d->start, d->stop);
	d->totalTime += elapsedTime;
	d->frames++;
	printf("\rAverage Time per frame: %5.1f ms", d->totalTime / d->frames);
}

void AnimExit(DataBlock* d)
{
	hipUnbindTexture(texInSrc);
	hipUnbindTexture(texOutSrc);
	hipUnbindTexture(texConstSrc);

	hipFree(d->d_InSrc);
	hipFree(d->d_OutSrc);
	hipFree(d->d_ConstSrc);

	hipEventDestroy(d->start);
	hipEventDestroy(d->stop);
}

int main()
{
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;

	hipEventCreate(&data.start);
	hipEventCreate(&data.stop);

	size_t imageSize = bitmap.image_size();

	hipMalloc((void**)&data.outBitmap,  imageSize);
	// assume float == 4 chars in size (i.e., rgba)
	hipMalloc((void**)&data.d_InSrc,    imageSize);
	hipMalloc((void**)&data.d_OutSrc,   imageSize);
	hipMalloc((void**)&data.d_ConstSrc, imageSize);

	// bind texture references to buffers
	hipBindTexture(NULL, texConstSrc, data.d_ConstSrc, imageSize);
	hipBindTexture(NULL, texInSrc,    data.d_InSrc,    imageSize);
	hipBindTexture(NULL, texOutSrc,   data.d_OutSrc,   imageSize);

	// initialize the constant data
	auto temp = (float*)malloc(imageSize);
	for (int i = 0; i < DIM * DIM; i++)
	{
		temp[i] = 0;
		int32_t x = i % DIM;
		int32_t y = i / DIM;
		if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
			temp[i] = MAX_TEMP;
	}
	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;
	for (int y = 800; y < 900; y++)
	{
		for (int x = 400; x < 500; x++)
			temp[x + y * DIM] = MIN_TEMP;
	}
	hipMemcpy(data.d_ConstSrc, temp, imageSize, hipMemcpyHostToDevice);

	// initialize the input data
	for (int y = 800; y < DIM; y++)
	{
		for (int x = 0; x < 200; x++)
			temp[x + y * DIM] = MAX_TEMP;
	}
	hipMemcpy(data.d_InSrc, temp, imageSize, hipMemcpyHostToDevice);
	free(temp);

	bitmap.anim_and_exit((void (*)(void*, int))AnimGPU, (void (*)(void*))AnimExit);

	return 0;
}
