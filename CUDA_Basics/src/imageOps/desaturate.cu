#include "hip/hip_runtime.h"
#include <cstdint>

#include <hip/hip_runtime.h>
#include <>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

#define cudaCheckError(val) CheckCuda(val, #val, __FILE__, __LINE__)

inline void CheckCuda(hipError_t result, const char* func, const char* file, const int32_t line)
{
	if (result)
	{
		printf("CUDA::ERROR_CODE::%d in function:'%s' file: '%s' line: %d\nERROR_DESCRIPTION:: %s\n", 
			result, func, file, line, hipGetErrorString(result));
		exit(-1);
	}
}

__global__ void Kernel(const uint8_t* inImg, uint8_t* outImg, int32_t width, int32_t height, int32_t channels)
{
	int32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	int32_t y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	int32_t idx = (x + y * width) * channels;

	outImg[idx + 0] = uint8_t(0.33f * inImg[idx + 0]);
	outImg[idx + 1] = uint8_t(0.33f * inImg[idx + 1]);
	outImg[idx + 2] = uint8_t(0.33f * inImg[idx + 2]);
	outImg[idx + 3] = inImg[idx + 3];
}


int main()
{
	constexpr uint32_t tx = 16;
	constexpr uint32_t ty = 16;

	int32_t  width, height, channels;
	uint8_t* inImg, * d_InImg, * outImg, * d_OutImg;

	stbi_set_flip_vertically_on_load(true);
	inImg = stbi_load("outImage/input.png", &width, &height, &channels, 0);
	const int32_t imgSize = width * height * channels;
	printf("Image info:\n\twidth = %d, height = %d, channels = %d\n",
		width, height, channels);

	cudaCheckError(hipMalloc((void**)&d_InImg,  imgSize * sizeof(uint8_t)));
	cudaCheckError(hipMalloc((void**)&d_OutImg, imgSize * sizeof(uint8_t)));
	outImg = (uint8_t*)malloc(imgSize * sizeof(uint8_t));
	
	cudaCheckError(hipMemcpy(d_InImg, inImg, imgSize * sizeof(uint8_t), hipMemcpyHostToDevice));

	dim3 threads(tx, ty);
	dim3 blocks((tx + width - 1) / tx, (ty + height - 1) / ty);
	printf("Kernel info:\n\tthreads = (%d, %d), blocks = (%d, %d)\n", 
		threads.x, threads.y, blocks.x, blocks.y);
	Kernel<<<blocks, threads>>>(d_InImg, d_OutImg, width, height, channels);
	cudaCheckError(hipGetLastError());

	cudaCheckError(hipMemcpy(outImg, d_OutImg, imgSize * sizeof(uint8_t), hipMemcpyDeviceToHost));

	// for (int i = 0; i < imgSize; i++)
	// 	printf("%3d\n", img[i]);

	stbi_flip_vertically_on_write(true);
	stbi_write_png("outImage/output.png", width, height, channels, outImg, width * channels);
	printf("Completed!\n");

	cudaCheckError(hipFree(d_InImg));
	cudaCheckError(hipFree(d_OutImg));
	free(inImg);
	free(outImg);
}
