#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>

#include <iostream>
#include <cassert>

// cuBLAS operates on matrix in col major

void VerifyResult(float* a, float* b, float* c, const int32_t N);
void VerifyResult(float* a, float* b, float* c, const int32_t M, const int32_t N, const int32_t K);

int main()
{
	constexpr int32_t N = 1 << 10;
	size_t bytes = N * N * sizeof(float);

	float* h_A = (float*)malloc(bytes);
	float* h_B = (float*)malloc(bytes);
	float* h_C = (float*)malloc(bytes);

	float* d_A, * d_B, * d_C;
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);

	// Pseudo Random Number Generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MT19937);

	// fill matrix with random values in device
	hiprandGenerateUniform(prng, d_A, N * N);
	hiprandGenerateUniform(prng, d_B, N * N);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const float alpha = 1.0f;
	const float beta  = 0.0f;

	// Calculate: c = (alpha * a) * b + (beta * c)
	// MxN = MxK * KxN 
	// Signature: handle, operation, operation,   M, N, K, alpha,   A, lda, B, ldb, beta, C, ldc
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

	// copy back to host 
	hipMemcpy(h_A, d_A, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_B, d_B, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

	VerifyResult(h_A, h_B, h_C, N);
	std::cout << "Completed Successfully!" << std::endl;

	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipblasDestroy(handle);

	std::cin.get();
	return 0;
}

void VerifyResult(float* a, float* b, float* c, const int32_t N)
{
	float temp;
	const float epsilon = 0.001;
	for (int row = 0; row < N; row++)
	{
		for (int col = 0; col < N; col++)
		{
			temp = 0;
			for (int i = 0; i < N; i++)
				temp += a[i * N + row] * b[col * N + i]; // col major
			assert(std::fabs(c[col * N + row] - temp) <= epsilon);
		}
	}
}


void VerifyResult(float* a, float* b, float* c, const int32_t M, const int32_t N, const int32_t K)
{
	float epsilon = 0.001; // Tolerance for our result
	for (int row = 0; row < M; row++)
	{
		for (int col = 0; col < N; col++)
		{
			float temp = 0.0f;
			for (int i = 0; i < K; i++)
				temp += a[i * M + row] * b[col * K + i];	// assuming col maj
			assert(std::fabs(c[col * M + row] - temp) <= epsilon);
		}
	}
}
